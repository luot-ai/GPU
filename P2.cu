#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc P2.cu -o P2 -I./src/submodule -Xcompiler "-O3 -std=c++14" -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_70,code=sm_70 -lhdf5_cpp -lhdf5

#include <random>
#include <iostream>
#include <vector>
#include <cfloat>
#include <cmath>
#include <chrono>
#include <iomanip>
#include <string>
#include <fstream>
#include <map>
#include <dirent.h>
#include <cstring>
#include <hdf5/serial/H5Cpp.h>
#include "Conv1d.hpp"
#include "BatchNorm1d.hpp"
#include "Linear.hpp"
#include "ReLU.hpp"
#include "Bmm.hpp"
#include "compare.hpp"
#include "usage.hpp"
#include <hip/hip_runtime.h>


/****************************************************************************************
 * 读取模型参数
 ****************************************************************************************/
// 获取目录中的所有 .txt 文件
std::vector<std::string> get_files_in_directory(const std::string& dir) {
    std::vector<std::string> files;
    DIR* dp;
    struct dirent* entry;
    if ((dp = opendir(dir.c_str())) != NULL) {
        while ((entry = readdir(dp)) != NULL) {
            std::string filename = entry->d_name;
            if (filename.find(".txt") != std::string::npos) {
                files.push_back(filename);
            }
        }
        closedir(dp);
    } else {
        perror("opendir");
    }
    return files;
}

// 读取 .txt 文件并转换为 std::vector<float>
std::map<std::string, std::vector<float>> params;
std::vector<float> read_param(const std::string& filepath) {
    std::vector<float> data;
    std::ifstream file(filepath);
    if (file.is_open()) {
        float value;
        while (file >> value) {
            data.push_back(value);
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filepath << std::endl;
    }
    return data;
}
void read_params(std::string dir) {
    // std::string dir = "."; // 当前目录

    // 获取目录中的所有 .txt 文件
    std::vector<std::string> param_files = get_files_in_directory(dir);
    for (const auto& file : param_files) {
        std::string filename = file.substr(0, file.find_last_of(".")); // 获取不带扩展名的文件名
        params[filename] = read_param(dir + "/" + file);
    }

    // // 访问参数时可以使用 params["conv1_weight"]
    // for (const auto& kv : params) {
    //     std::cout << "Key: " << kv.first << ", Values: ";
    //     // for (const auto& value : kv.second) {
    //     //     std::cout << value << " ";
    //     // }
    //     std::cout << std::endl;
    // }

    return ;
}

struct fcp {
    float* weight; // Conv weight
    float* bias;   // Conv bias
};
void read_fcp(const std::string& layer, fcp& wbp,int i) {
    std::string fiStr = std::to_string(i);;
    std::string name = layer + "fc" + fiStr;  
    //std::cout << name << std::endl;
    hipMalloc((void**)&wbp.weight, params[name + ".weight"].size() * sizeof(float));
    hipMalloc((void**)&wbp.bias, params[name + ".bias"].size() * sizeof(float));
    hipMemcpy(wbp.weight, params[name + ".weight"].data(), params[name + ".weight"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbp.bias, params[name + ".bias"].data(), params[name + ".bias"].size() * sizeof(float), hipMemcpyHostToDevice);
}
void free_fcp(fcp& wbp){
    hipFree(wbp.bias);
    hipFree(wbp.weight);
}

struct wbBnP {
    float* weight; // Conv weight
    float* bias;   // Conv bias
    float* bn_weight; // BatchNorm weight
    float* bn_bias;   // BatchNorm bias
    float* bn_mean;   // BatchNorm running mean
    float* bn_var;    // BatchNorm running var
};
void read_wbBnP(const std::string& layer,const std::string& cf,wbBnP& wbBnP,int i,int param_offset=0) {

    std::string cfiStr = std::to_string(i);
    std::string biStr = std::to_string(i+param_offset);
    std::string name = layer + cf + cfiStr;
    std::string bnStr = layer + "bn" + biStr;   
    //std::cout << name << std::endl;
    //std::cout << bnStr << std::endl;
    hipMalloc((void**)&wbBnP.weight, params[name + ".weight"].size() * sizeof(float));
    hipMalloc((void**)&wbBnP.bias, params[name + ".bias"].size() * sizeof(float));
    hipMalloc((void**)&wbBnP.bn_weight, params[bnStr + ".weight"].size() * sizeof(float));
    hipMalloc((void**)&wbBnP.bn_bias, params[bnStr + ".bias"].size() * sizeof(float));
    hipMalloc((void**)&wbBnP.bn_mean, params[bnStr + ".running_mean"].size() * sizeof(float));
    hipMalloc((void**)&wbBnP.bn_var, params[bnStr + ".running_var"].size() * sizeof(float));
    hipMemcpy(wbBnP.weight, params[name + ".weight"].data(), params[name + ".weight"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbBnP.bias, params[name + ".bias"].data(), params[name + ".bias"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbBnP.bn_weight, params[bnStr + ".weight"].data(), params[bnStr + ".weight"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbBnP.bn_bias, params[bnStr + ".bias"].data(), params[bnStr + ".bias"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbBnP.bn_mean, params[bnStr + ".running_mean"].data(), params[bnStr + ".running_mean"].size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wbBnP.bn_var, params[bnStr + ".running_var"].data(), params[bnStr + ".running_var"].size() * sizeof(float), hipMemcpyHostToDevice);
}
void free_wbBnP(wbBnP& wbBnP){
    hipFree(wbBnP.bias);
    hipFree(wbBnP.weight);
    hipFree(wbBnP.bn_bias);
    hipFree(wbBnP.bn_mean);
    hipFree(wbBnP.bn_var);
    hipFree(wbBnP.bn_weight);
}

struct CB3P {
    wbBnP cb1;
    wbBnP cb2;
    wbBnP cb3;
};
void read_CB3P(const std::string& layer,CB3P& CB3P) {
    read_wbBnP(layer,"conv",CB3P.cb1,1);
    read_wbBnP(layer,"conv",CB3P.cb2,2);
    read_wbBnP(layer,"conv",CB3P.cb3,3);   
}
void free_CB3P(CB3P &CB3P){
    free_wbBnP(CB3P.cb1);
    free_wbBnP(CB3P.cb2);
    free_wbBnP(CB3P.cb3);
}


struct FB2FP {
    wbBnP fb1;
    wbBnP fb2;
    fcp   f3;
};
void read_FB2FP(const std::string& layer,FB2FP& FB2FP,int param_offset=0)    {
    read_wbBnP(layer,"fc",FB2FP.fb1,1,param_offset);
    read_wbBnP(layer,"fc",FB2FP.fb2,2,param_offset);
    read_fcp(layer,FB2FP.f3,3);
}
void free_FB2FP(FB2FP &FB2FP){
    free_wbBnP(FB2FP.fb1);
    free_wbBnP(FB2FP.fb2);
    free_fcp(FB2FP.f3);
}

struct stndP {
    CB3P cb3;
    FB2FP fb2f;
};
void read_stndP(const std::string& layer,stndP& stndP) {
    read_CB3P(layer,stndP.cb3);
    read_FB2FP(layer,stndP.fb2f,3);
}
void free_stndP(stndP& stndP){
    free_CB3P(stndP.cb3);
    free_FB2FP(stndP.fb2f);
}

struct cudaP {
    stndP stn3dp;
    stndP stnkdp;
    CB3P  featp;
    FB2FP nonep;
};
void freeDP(cudaP &dp)
{
    free_stndP(dp.stn3dp);
    free_stndP(dp.stnkdp);
    free_CB3P(dp.featp);
    free_FB2FP(dp.nonep);
}

cudaP dParams;


/****************************************************************************************
 * 读取训练集数据
 ****************************************************************************************/

using namespace H5;
void read_h5_file(const std::string& file_path, std::vector<std::vector<float>>& list_of_points, std::vector<int>& list_of_labels) {
    try {
        // 打开文件
        H5File file(file_path, H5F_ACC_RDONLY);

        // 获取文件中的所有数据集名称
        std::vector<std::string> dataset_names;
        hsize_t num_objs = file.getNumObjs();
        for (hsize_t i = 0; i < num_objs; i++) {
            dataset_names.push_back(file.getObjnameByIdx(i));
        }

        // 读取每个数据集
        for (const auto& name : dataset_names) {
            DataSet dataset = file.openDataSet(name + "/points");
            DataSpace dataspace = dataset.getSpace();

            // 获取数据集的维度
            hsize_t dims[2];
            dataspace.getSimpleExtentDims(dims, NULL);

            // 读取数据
            std::vector<float> points(dims[0] * dims[1]);
            dataset.read(points.data(), PredType::NATIVE_FLOAT);

            // 存储点云数据
            list_of_points.push_back(points);

            // 读取标签
            Attribute label_attr = file.openGroup(name).openAttribute("label");
            int label;
            label_attr.read(PredType::NATIVE_INT, &label);

            // 存储标签
            list_of_labels.push_back(label);
        }
    } catch (FileIException& error) {
        error.printErrorStack();
    } catch (DataSetIException& error) {
        error.printErrorStack();
    } catch (DataSpaceIException& error) {
        error.printErrorStack();
    } catch (DataTypeIException& error) {
        error.printErrorStack();
    }
}





/****************************************************************************************
 * 网络搭建
 ****************************************************************************************/
__global__ void LogSoftMax_Kernel(float* input,float* output,int L,int BatchSize = 32)
{
    int bx = blockIdx.x;
    int index = bx;
    if (index < gridDim.x)
    {
        float sum = 0;
        for (int l = 0; l < L; l++)
        {
            int iIdx = l + index * L;
            input[iIdx] = exp(input[iIdx]);
            sum += input[iIdx];
        }
        for (int l = 0; l < L; l++)
        {
            int iIdx = l + index * L;
            output[iIdx] = log(input[iIdx] / sum);
        }
    }
}
void LogSoftMax_GPU(float* input,float* output,int L,int BatchSize = 32)
{   
    dim3 blockDim(1);
    dim3 gridDim(BatchSize);
    LogSoftMax_Kernel<<<gridDim,blockDim>>>(input,output,L,BatchSize);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void matrix_add_I_kernel(float *input, int n)
{
    int i = threadIdx.x;
    input[i * n + i] = input[i * n + i] + 1.0f;
}
__global__ void matrix_add_I_kernel_normal(float *input, int n,int batchSize)
{
    int curN = blockIdx.x;
    int curB = threadIdx.x;
    int index = curB* gridDim.x + curN;
    int iidx =  index*n +curN;
    if (index < n * batchSize )
        input[iidx] = input[iidx] + 1.0f;
}
void matrix_add_I(float *input, int n,int batchSize)
{
    // if (n <= 1024)
    // {
    //     dim3 blockDim(n);
    //     dim3 gridDim(batchSize);
    //     matrix_add_I_kernel<<<gridDim, blockDim>>>(input, n,batchSize);
    // }
    // else
    // {
        dim3 blockDim(batchSize);
        dim3 gridDim(n);
        matrix_add_I_kernel_normal<<<gridDim, blockDim>>>(input, n,batchSize);
    hipDeviceSynchronize();
}


__global__ void Maxpooling_Kernel(float* input,float* output,int numPoints)
{
    __shared__ float sharedMax[1024];
    
    int tx = threadIdx.x;
    int channel = blockIdx.x;

    float localMax = -FLT_MAX;
    int cnum = channel * numPoints;
    for (int i = tx; i < numPoints; i += blockDim.x) {
        float val = input[cnum + i];
        if (val > localMax) {
            localMax = val;
        }
    }
    sharedMax[tx] = localMax;
    __syncthreads();

    // 归约：逐步计算块内的最大值
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            if (sharedMax[tx + stride] > sharedMax[tx]) {
                sharedMax[tx] = sharedMax[tx + stride];
            }
        }
        __syncthreads();
    }

    // 线程0写入最终的最大值
    if (tx == 0) {
        output[channel] = sharedMax[0];
    }
}
void GPU_MaxPooling(int ics, int batchSize, int numPoints,float* input, float* output)
{
    std::cout << "----START MAXPOOLING" << std::endl;
    dim3 gridDim(ics*batchSize);
    dim3 blockDim(1024);
    Maxpooling_Kernel<<<gridDim, blockDim>>>(input, output,numPoints);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}
__global__ void BMM_Kernel(float* input_A,float* input_B,float* output,int M_A,int K_A,int K_B,int N_B,int BatchSize)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    //int bz = blockIdx.z;

    int col = tx + bx * blockDim.x;
    int row = ty + by * blockDim.y;
    int batch = blockIdx.z;

    if (row < M_A && col < N_B)
    {
        float tmp = 0.0f;
        for (int k =0;k<K_A;k++)
        {
            tmp += input_A[batch * M_A * K_A + row * K_A + k] * input_B[batch * K_B * N_B + k * N_B + col];
        }
        output[batch*M_A*N_B+row*N_B+col] = tmp;
    }
}
void GPU_Bmm(float* input_A,float* input_B,float* output,int M_A,int K_A,int K_B,int N_B,int BatchSize = 1)
{
    std::cout << "--------BMM" << std::endl;

    const int BLK_X = 32;
    const int BLK_Y = 32;

    dim3 blockDim(BLK_X, BLK_Y);
    dim3 gridDim((N_B + BLK_X - 1) / BLK_X, (M_A + BLK_Y - 1) / BLK_Y,BatchSize);//X:宽度 Y：高度
    BMM_Kernel<<<gridDim, blockDim>>>(input_A, input_B, output, M_A, K_A, K_B, N_B, BatchSize);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void transpose_Kernel(float* input,float* output,int dim0,int dim1,int dim2)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int idx = tx + bx * blockDim.x;
    int idy = ty + by * blockDim.y;
    int index = idx + idy * dim1;

    if (idx < dim1 && idy < dim2)
    {
        for (int b=0;b<dim0;b++)
        {
            int bdd = b*dim1*dim2;
            output[bdd+index]=input[bdd+idx*dim2+idy];
        }
    }
}
void GPU_transpose(float* input,float* output,int dim0,int dim1,int dim2)
{
    const int BLK_X = 32;
    const int BLK_Y = 32;

    dim3 blockDim(BLK_X, BLK_Y);
    dim3 gridDim((dim1 + BLK_X -1)/BLK_X,  (dim2+BLK_Y-1)/BLK_Y);
    transpose_Kernel<<<gridDim, blockDim>>>(input,output,dim0,dim1,dim2);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void linear_Kernel(int inFeatures,float* weight,float* bias,float* input,float* output,int outFeatures,int bacthSize)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int curOC = tx + bx * blockDim.x;
    int curB = ty + by * blockDim.y;

    int index = curOC + curB * outFeatures;
    if (curOC < outFeatures && curB < bacthSize)
    {
        output[index] = bias[curOC];
        for (int ic = 0; ic < inFeatures; ic++)
        {
            output[index] +=
                input[curB * inFeatures + ic] *
                weight[curOC * inFeatures + ic];
        }
    }
}
void Linear_GPU(int batchSize,int inFeatures, int outFeatures,float* cudaWeights,float* cudaBias,float* input,float* output){
    std::cout << "------------LAYER:linear" << std::endl;
    dim3 blockDim(32,32);
    dim3 gridDim((outFeatures+31)/32,(batchSize+31)/32);
    linear_Kernel<<<gridDim,blockDim>>>(inFeatures,cudaWeights,cudaBias,input,output,outFeatures,batchSize);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());
    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void ReLu_Kernel(float *input,float *output)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    int idx = tx + bx * blockDim.x;
    int index = idx ;

    output[index] = input[index] > 0 ? input[index] : 0;

}
void ReLU_GPU(int batchSize,int numPoints,int OC,float* input,float* output){
    std::cout << "------------LAYER:relu" << std::endl;
    // const int BLK_X = 32;
    // const int BLK_Y = 32;

    dim3 blockDim(OC);
    dim3 gridDim(batchSize*numPoints);
    ReLu_Kernel<<<gridDim, blockDim>>>(input, output);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void BatchNorm1d_Kernel(int numPoints,float* weight,float* bias,float* running_mean,float* running_var,float* input,float* output,float esp = 1e-5)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int idx = tx + bx * blockDim.x;
    int index = idx;

    if (idx < blockDim.x * gridDim.x)
    {
        float mean = running_mean[tx];
        float var = running_var[tx];
        for (int n = 0; n < numPoints; n++)
        {
            int iIdx = index * numPoints + n;
            output[iIdx] = (input[iIdx] - mean) / sqrt(var + esp) * weight[tx] + bias[tx];
        }
    }
}
void BatchNorm1d_GPU(int numFeatures, int batchSize, int numPoints,float* weight,float* bias,float* running_mean,float* running_var,float* input,float* output,float esp = 1e-5)
{
    float* cudaWeights;
    float* cudaBias;
    float* cudaRV;
    float* cudaRM;

    hipMalloc((void **)&cudaWeights, numFeatures * sizeof(float));
    hipMalloc((void **)&cudaBias, numFeatures * sizeof(float));
    hipMalloc((void **)&cudaRV, numFeatures * sizeof(float));
    hipMalloc((void **)&cudaRM, numFeatures * sizeof(float));

    hipMemcpy(cudaWeights, weight, numFeatures * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaBias, bias, numFeatures * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaRV, running_var, numFeatures * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaRM, running_mean, numFeatures * sizeof(float), hipMemcpyHostToDevice);

    std::cout << "------------LAYER:batchnorm" << std::endl;
    dim3 blockDim(numFeatures);
    dim3 gridDim(batchSize);
    BatchNorm1d_Kernel<<<gridDim, blockDim>>>(numPoints,cudaWeights,cudaBias,cudaRM,cudaRV,input,output);

    hipFree(cudaWeights);
    hipFree(cudaBias);
    hipFree(cudaRV);
    hipFree(cudaRM);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void Conv1d_Kernel(int outChannels,int batchSize,int numPoints,int inChannels,float* input, float* weights, float* bias, float* output)
{
    int oc = threadIdx.x;
    int b = blockIdx.x;
    int index = oc + b * blockDim.x;
    //printf("oc %d, batch %d, index %d\n",oc,b, index);
    if(index >= outChannels * batchSize)
        return ;
    for (int n=0;n<numPoints;n++)
    {
        //printf("numpoint: %d\n",n);
        float res = bias[oc];
        //printf("the res of index %d is : %f\n",index*numPoints+n,res);
        for (int ic=0;ic<inChannels;ic++ )
        {
            int ii = b*inChannels*numPoints+ic*numPoints+n;
            int ww = oc*inChannels+ic;
            //printf("input: %d,weight: %d\n",ii,ww);
            res += input[ii]*weights[ww];
            //printf("the res of index %d is : %f\n",index*numPoints+n,res);
        }
        output[index*numPoints+n]=res;
    }
    // hipFree(w);
}
void Conv1d_GPU(int batchSize,int numPoints,int inChannels,int outChannels,int kSize,float* input, float* weights, float* bias, float* output ){
    //int L=numPoints;
    std::cout << "------------LAYER:convolution" << std::endl;

    float* cudaWeights;
    float* cudaBias;
    hipMalloc((void **)&cudaWeights, inChannels * outChannels * sizeof(float));
    hipMalloc((void **)&cudaBias, outChannels * sizeof(float));
    hipMemcpy(cudaWeights, weights, inChannels * outChannels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaBias, bias, outChannels * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(outChannels);
    dim3 gridDim(batchSize);
    //std::cout << "WIDTH: " << numPoints << ", IC: " << inChannels << ", OC: " << outChannels << std::endl;
    //std::cout << "isize: " << input.size() << ", wsize: " << weights.size() << ", bsize: " << bias.size() << ", osize: " << output.size() << std::endl;
    Conv1d_Kernel<<<gridDim,blockDim>>>(outChannels,batchSize,numPoints,inChannels,input,cudaWeights,cudaBias,output);
    //printVector_GPU(output,batchSize*numPoints*outChannels);
    hipFree(cudaWeights);
    hipFree(cudaBias);
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
    //printVector_GPU(output,batchSize*numPoints*outChannels);
}

//ARCH CB
template<int TILEX,int TILEY>
__global__ void CBWRAP_Kernel(int outChannels,int batchSize,int numPoints,int inChannels,float* input, 
float* convWeights, float* convBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int np = tx + bx * blockDim.x;
    int oc = ty + by * blockDim.y;
    int b = blockIdx.z;
    //printf("KERNEL: inchannel %d, outchannel %d, numPoints %d\n",inChannels,outChannels,numPoints);
    // if(oc >= outChannels || np >= numPoints)
    //     return ;
    
    __shared__ float ds_weights[TILEX][TILEY];
    __shared__ float ds_input[TILEX][TILEY];
    // __shared__ float ds_bias[TILEY];
    // __shared__ float ds_bnRM[TILEY];
    // __shared__ float ds_bnRV[TILEY];
    // __shared__ float ds_bnB[TILEY];
    // __shared__ float ds_bnW[TILEY];
    // __shared__ float ds_res[TILEY];

    //phases
    float mean = bnRM[oc];
    float var = bnRV[oc];
    float bnW = bnWeights[oc];
    float bnB = bnBias[oc];
    float res = convBias[oc];
    for (int i = 0; i < inChannels / TILEX; ++i)
    {
        // loading input and weights
        ds_weights[ty][tx] = convWeights[oc * inChannels + i * TILEX + tx];
        if (np < numPoints)
        {
            ds_input[tx][ty] = input[b * numPoints * inChannels + (i * TILEY + ty) * numPoints + np];
        }
        else
        {
            ds_input[tx][ty] = 0;
        }
        __syncthreads();
        // calculate:iterations
        for (int j = 0; j < TILEX; ++j)
        {
            res += ds_weights[ty][j] * ds_input[tx][j];
        }
        __syncthreads();
    }
    res = (res - mean) / sqrt(var + esp) * bnW + bnB;
    if(np < numPoints)
        output[b * numPoints * outChannels + oc * numPoints + np] = res;
}

template<int TILEX,int TILEY>
__global__ void CBWRAP_Kernel_np4tms(int outChannels,int batchSize,int numPoints,int inChannels,float* input, 
float* convWeights, float* convBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int np = tx + bx * blockDim.x;
    int oc = ty + by * blockDim.y;
    int b = blockIdx.z;
    //printf("KERNEL: inchannel %d, outchannel %d, numPoints %d\n",inChannels,outChannels,numPoints);
    // if(oc >= outChannels || np >= numPoints)
    //     return ;
    
    __shared__ float ds_weights[TILEX][TILEY];
    __shared__ float ds_input[TILEX][TILEY];
    // __shared__ float ds_bias[TILEY];
    // __shared__ float ds_bnRM[TILEY];
    // __shared__ float ds_bnRV[TILEY];
    // __shared__ float ds_bnB[TILEY];
    // __shared__ float ds_bnW[TILEY];
    // __shared__ float ds_res[TILEY];

    //phases
    float mean = bnRM[oc];
    float var = bnRV[oc];
    float bnW = bnWeights[oc];
    float bnB = bnBias[oc];
    float res = convBias[oc];
    for (int i = 0; i < inChannels / TILEX; ++i)
    {
        // loading input and weights
        ds_weights[ty][tx] = convWeights[oc * inChannels + i * TILEX + tx];
        ds_input[tx][ty] = input[b * numPoints * inChannels + (i * TILEY + ty) * numPoints + np];
        __syncthreads();
        // calculate:iterations
        for (int j = 0; j < TILEX; ++j)
        {
            res += ds_weights[ty][j] * ds_input[tx][j];
        }
        __syncthreads();
    }
    res = (res - mean) / sqrt(var + esp) * bnW + bnB;
    output[b * numPoints * outChannels + oc * numPoints + np] = res;
}


void CBWRAP_GPU(int batchSize,int numPoints,int inChannels,int outChannels,int kSize,float* input, 
float* cudaConvWeights, float* cudaConvBias, 
float* cudaBnWeights,float* cudaBnBias,float* cudaBnRM,float* cudaBnRV,float* output,float esp = 1e-5
){
    std::cout << "------------LAYER:CBWRAP" << std::endl;
    const int BLK_X = 8;
    const int BLK_Y = 8;
    dim3 blockDim(BLK_X, BLK_Y);
    dim3 gridDim((numPoints + BLK_X - 1) / BLK_X, (outChannels + BLK_Y - 1) / BLK_Y, batchSize); // X:宽度 Y：高度

    if (numPoints % BLK_X == 0)
    {
        printf("hey!!\n");
        CBWRAP_Kernel_np4tms<BLK_X, BLK_Y><<<gridDim, blockDim>>>(outChannels, batchSize, numPoints, inChannels, input, cudaConvWeights, cudaConvBias,
                                                                  cudaBnWeights, cudaBnBias, cudaBnRM, cudaBnRV, output);
    }
    else
    {
        CBWRAP_Kernel<BLK_X, BLK_Y><<<gridDim, blockDim>>>(outChannels, batchSize, numPoints, inChannels, input, cudaConvWeights, cudaConvBias,
                                                           cudaBnWeights, cudaBnBias, cudaBnRM, cudaBnRV, output);
    }

    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());

    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}


// ARCH CBR
template<int TILEX,int TILEY>
__global__ void CBRWRAP_Kernel(int outChannels,int batchSize,int numPoints,int inChannels,float* input, 
float* convWeights, float* convBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int np = tx + bx * blockDim.x;
    int oc = ty + by * blockDim.y;
    int b = blockIdx.z;
    //printf("KERNEL: inchannel %d, outchannel %d, numPoints %d\n",inChannels,outChannels,numPoints);
    // if(oc >= outChannels || np >= numPoints)
    //     return ;
    
    __shared__ float ds_weights[TILEX][TILEY];
    __shared__ float ds_input[TILEX][TILEY];
    // __shared__ float ds_bias[TILEY];
    // __shared__ float ds_bnRM[TILEY];
    // __shared__ float ds_bnRV[TILEY];
    // __shared__ float ds_bnB[TILEY];
    // __shared__ float ds_bnW[TILEY];
    // __shared__ float ds_res[TILEY];

    //phases
    float mean = bnRM[oc];
    float var = bnRV[oc];
    float bnW = bnWeights[oc];
    float bnB = bnBias[oc];
    float res = convBias[oc];
    for (int i = 0; i < inChannels / TILEX; ++i)
    {
        // loading input and weights
        ds_weights[ty][tx] = convWeights[oc * inChannels + i * TILEX + tx];
        if (np < numPoints)
        {
            ds_input[tx][ty] = input[b * numPoints * inChannels + (i * TILEY + ty) * numPoints + np];
        }
        else
        {
            ds_input[tx][ty] = 0;
        }
        __syncthreads();
        // calculate:iterations
        for (int j = 0; j < TILEX; ++j)
        {
            res += ds_weights[ty][j] * ds_input[tx][j];
        }
        __syncthreads();
    }
    res = (res - mean) / sqrt(var + esp) * bnW + bnB;
    if (res < 0)
        res = 0;
    if(np < numPoints)
        output[b * numPoints * outChannels + oc * numPoints + np] = res;
}

template<int TILEX,int TILEY>
__global__ void CBRWRAP_Kernel_np4tms(int outChannels,int batchSize,int numPoints,int inChannels,float* input, 
float* convWeights, float* convBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int np = tx + bx * blockDim.x;
    int oc = ty + by * blockDim.y;
    int b = blockIdx.z;
    //printf("KERNEL: inchannel %d, outchannel %d, numPoints %d\n",inChannels,outChannels,numPoints);
    // if(oc >= outChannels || np >= numPoints)
    //     return ;
    
    __shared__ float ds_weights[TILEX][TILEY];
    __shared__ float ds_input[TILEX][TILEY];
    // __shared__ float ds_bias[TILEY];
    // __shared__ float ds_bnRM[TILEY];
    // __shared__ float ds_bnRV[TILEY];
    // __shared__ float ds_bnB[TILEY];
    // __shared__ float ds_bnW[TILEY];
    // __shared__ float ds_res[TILEY];

    //phases
    float mean = bnRM[oc];
    float var = bnRV[oc];
    float bnW = bnWeights[oc];
    float bnB = bnBias[oc];
    float res = convBias[oc];
    for (int i = 0; i < inChannels / TILEX; ++i)
    {
        // loading input and weights
        ds_weights[ty][tx] = convWeights[oc * inChannels + i * TILEX + tx];
        ds_input[tx][ty] = input[b * numPoints * inChannels + (i * TILEY + ty) * numPoints + np];
        __syncthreads();
        // calculate:iterations
        for (int j = 0; j < TILEX; ++j)
        {
            res += ds_weights[ty][j] * ds_input[tx][j];
        }
        __syncthreads();
    }
    res = (res - mean) / sqrt(var + esp) * bnW + bnB;
    if (res < 0)
        res = 0;
    output[b * numPoints * outChannels + oc * numPoints + np] = res;
}

__global__ void CBRWRAP_Kernel_ic3(int TILEX,int TILEY,int outChannels,int batchSize,int numPoints,int inChannels,float* input, 
float* convWeights, float* convBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int np = tx + bx * blockDim.x;
    int oc = ty + by * blockDim.y;
    int b = blockIdx.z;
    //printf("oc %d, batch %d, index %d\n",oc,b, index);
    if(oc >= outChannels || np >= numPoints)
        return ;
    
    float mean = bnRM[oc];
    float var = bnRV[oc];
    float bnW = bnWeights[oc];
    float bnB = bnBias[oc];
    float res = convBias[oc];

    for (int ic = 0; ic < inChannels; ic++)
    {
        int ii = b * inChannels * numPoints + ic * numPoints + np;
        int ww = oc * inChannels + ic;
        res += input[ii] * convWeights[ww];
    }
    res = (res - mean) / sqrt(var + esp) * bnW + bnB;
    res = res > 0 ? res : 0;
    output[b * numPoints * outChannels + oc * numPoints + np] = res;
}
void CBRWRAP_GPU(int batchSize,int numPoints,int inChannels,int outChannels,int kSize,float* input, 
float* cudaConvWeights, float* cudaConvBias, 
float* cudaBnWeights,float* cudaBnBias,float* cudaBnRM,float* cudaBnRV,float* output,float esp = 1e-5
){
    std::cout << "------------LAYER:CBRWRAP" << std::endl;
    // printf("inchannel %d,numPoints %d\n",inChannels,numPoints);

    const int BLK_X = 8;
    const int BLK_Y = 8;
    dim3 blockDim(BLK_X,BLK_Y);
    //dim3 gridDim((numPoints + BLK_X - 1) / BLK_X,(outChannels + BLK_Y - 1) / BLK_Y);//X:宽度 Y：高度
    dim3 gridDim((numPoints + BLK_X - 1) / BLK_X,(outChannels + BLK_Y - 1) / BLK_Y,batchSize);//X:宽度 Y：高度

    //std::cout << "WIDTH: " << numPoints << ", IC: " << inChannels << ", OC: " << outChannels << std::endl;
    //std::cout << "isize: " << input.size() << ", wsize: " << weights.size() << ", bsize: " << bias.size() << ", osize: " << output.size() << std::endl;
    if (inChannels == 3)
    {
        CBRWRAP_Kernel_ic3<<<gridDim, blockDim>>>(BLK_X, BLK_Y, outChannels, batchSize, numPoints, inChannels, input, cudaConvWeights, cudaConvBias,
                                                cudaBnWeights, cudaBnBias, cudaBnRM, cudaBnRV, output);
    }
    else
    {
        //printf("KERNEL: inchannel %d, outchannel %d, numPoints %d\n",inChannels,outChannels,numPoints);
        if (numPoints % BLK_X == 0)
        {
            printf("hey!!\n");
            CBRWRAP_Kernel_np4tms<BLK_X, BLK_Y><<<gridDim, blockDim>>>( outChannels, batchSize, numPoints, inChannels, input, cudaConvWeights, cudaConvBias,
                                              cudaBnWeights, cudaBnBias, cudaBnRM, cudaBnRV, output);
        }
        else
        {
            CBRWRAP_Kernel<BLK_X, BLK_Y><<<gridDim, blockDim>>>( outChannels, batchSize, numPoints, inChannels, input, cudaConvWeights, cudaConvBias,
                                              cudaBnWeights, cudaBnBias, cudaBnRM, cudaBnRV, output);
        }
    }
    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());
    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}

void GPU_CBR(int batchSize, int numPoints, int inics, int OC,wbBnP& wbBnP, float* input, float* reluOutput)
{
    CBRWRAP_GPU(batchSize,numPoints,inics,OC,1,input,wbBnP.weight,wbBnP.bias,
    wbBnP.bn_weight,wbBnP.bn_bias,wbBnP.bn_mean,wbBnP.bn_var,reluOutput);
}
void GPU_CBR_3 (int OC1,int OC2,int OC3,int batchSize,int numPoints,int inics,CB3P &cb3p, float* input, float* output) {
    std::cout << "----START CBR_3" << std::endl;
    int bn = batchSize * numPoints;
    float* relu1_output;
    float* relu2_output;
    
    hipMalloc((void **)&relu1_output, bn*OC1 * sizeof(float));
    hipMalloc((void **)&relu2_output, bn*OC2 * sizeof(float));

    GPU_CBR(batchSize, numPoints, inics, OC1, cb3p.cb1, input, relu1_output);
    GPU_CBR(batchSize, numPoints, OC1, OC2, cb3p.cb2, relu1_output, relu2_output);
    GPU_CBR(batchSize, numPoints, OC2, OC3, cb3p.cb3, relu2_output, output);
    //printVector_GPU(output, bn * OC3);
    hipFree(relu1_output);
    hipFree(relu2_output);
}


// ARCH FBR
__global__ void FBRWRAP_Kernel(int TILEX,int TILEY,int outFeatures,int batchSize,int inFeatures,float* input, 
float* fcWeights, float* fcBias, 
float* bnWeights,float* bnBias,float* bnRM,float* bnRV,float* output,float esp = 1e-5 )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int oc = tx + bx * blockDim.x;
    int curB = ty + by * blockDim.y;

    if (oc < outFeatures && curB < batchSize)
    {
        float mean = bnRM[oc];
        float var = bnRV[oc];
        float bnW = bnWeights[oc];
        float bnB = bnBias[oc];
        float res = fcBias[oc];
        for (int ic = 0; ic < inFeatures; ic++)
        {
            res +=
                input[curB * inFeatures + ic] *
                fcWeights[oc * inFeatures + ic];
        }
        res = (res - mean) / sqrt(var + esp) * bnW + bnB;
        res = res > 0 ? res : 0;
        int index = oc + curB * outFeatures;
        output[index] = res;
    }
}
void FBRWRAP_GPU(int batchSize,int inFeatures,int outFeatures,float* input, 
float* cudaFcWeights, float* cudaFcBias, 
float* cudaBnWeights,float* cudaBnBias,float* cudaBnRM,float* cudaBnRV,float* output,float esp = 1e-5
){
    std::cout << "------------LAYER:FBRWRAP" << std::endl;
    // printf("inchannel %d,numPoints %d\n",inChannels,numPoints);
    const int BLK_X = 32;
    const int BLK_Y = 32;
    dim3 blockDim(BLK_X,BLK_Y);
    dim3 gridDim((outFeatures + BLK_X - 1) / BLK_X,(batchSize + BLK_Y - 1) / BLK_Y);//X:宽度 Y：高度
    FBRWRAP_Kernel<<<gridDim,blockDim>>>(BLK_X,BLK_Y,outFeatures,batchSize,inFeatures,input,cudaFcWeights,cudaFcBias,cudaBnWeights,cudaBnBias,cudaBnRM,cudaBnRV,
    output);

    // 检查内核启动是否成功
    CUDA_CHECK(hipGetLastError());
    // 同步设备并检查执行错误
    CUDA_CHECK(hipDeviceSynchronize());
}
void GPU_FBR(int batchSize, int inFeatures, 
int outFeatures,wbBnP& fbp, float* input, float* reluOutput)
{
    FBRWRAP_GPU(batchSize,inFeatures,outFeatures,input,
    fbp.weight,fbp.bias,
    fbp.bn_weight,fbp.bn_bias,
    fbp.bn_mean,fbp.bn_var,reluOutput);
}
void GPU_FBR_2_F(int OC1,int OC2,int OC3,int batchSize,int inics,FB2FP &fb2f, float* input, float* output,int param_offset=3)
{
    std::cout << "----START FBR_2_F" << std::endl;
    float* relu1_output;
    float* relu2_output;

    hipMalloc((void **)&relu1_output, batchSize*OC1 * sizeof(float));
    hipMalloc((void **)&relu2_output, batchSize*OC2 * sizeof(float));

    GPU_FBR(batchSize,inics,OC1,fb2f.fb1,input,relu1_output);
    GPU_FBR(batchSize,OC1,OC2,fb2f.fb2,relu1_output,relu2_output);
    Linear_GPU(batchSize,OC2, OC3,fb2f.f3.weight, fb2f.f3.bias, relu2_output, output);

    hipFree(relu1_output);
    hipFree(relu2_output);
}



std::vector<int> Inference_GPU (int inChannels,
            int batchSize,
            int numPoints,float* input,float* output,
            float* stn3d_out,
            float* stnkd_out,
            const std::vector<float>& C1={},
            const std::vector<float>& C2={},
            const std::vector<float>& C3={},
            const std::vector<float>& C4={},
            bool compare=false) {

    //copyParamsToDevice();
    std::cout << "**********************START INFERENCE************************" << std::endl;
    std::cout << "PART1:STN3d" << std::endl;
    int bn = batchSize * numPoints;
    int OC1 = 64;
    int OC2 = 128;
    int OC3 = 1024;
    int FC_OC1 = 512;
    int FC_OC2 = 256;
    int FC_OC3 = 9;
    float* CBR3_output;
    float* maxp_output;
    hipMalloc((void **)&CBR3_output, bn * OC3 * sizeof(float));
    hipMalloc((void **)&maxp_output, batchSize * OC3 * sizeof(float));
    GPU_CBR_3(OC1,OC2,OC3, batchSize, numPoints,inChannels,dParams.stn3dp.cb3, input, CBR3_output);   // conv-bn-relu * 3
    GPU_MaxPooling(OC3, batchSize, numPoints,CBR3_output, maxp_output); // Max pooling    
    GPU_FBR_2_F(FC_OC1,FC_OC2,FC_OC3,batchSize,OC3,dParams.stn3dp.fb2f,maxp_output,stn3d_out);// fc-bn-relu * 2 + fc
    if (compare)
    {
        compareVectors_GPU(C1,CBR3_output,bn*OC3);
        compareVectors_GPU(C2,maxp_output,batchSize*OC3);
        compareVectors_GPU(C3,stn3d_out,batchSize*FC_OC3);
    }
    matrix_add_I(stn3d_out,3,batchSize);
    if(compare)
    {
        compareVectors_GPU(C4,stn3d_out,batchSize*FC_OC3);
        printVector(C3);
        printVector(C4);
    }

    
    hipFree(CBR3_output);
    hipFree(maxp_output);

    std::cout << "PART2:TRANS->BMM->TRANS->CBR" << std::endl;
    int encoderIC1 = inChannels;
    int fstn_inChannel = 64;//encoderOC1
    float* input_trans;
    float* bmm1_res;
    float* bmm1_res_trans;
    float* fstn_input;
    hipMalloc((void **)&input_trans, bn * inChannels * sizeof(float));
    hipMalloc((void **)&bmm1_res, batchSize*numPoints*encoderIC1 * sizeof(float));
    hipMalloc((void **)&bmm1_res_trans, batchSize*encoderIC1*numPoints * sizeof(float));
    hipMalloc((void **)&fstn_input, batchSize*fstn_inChannel*numPoints * sizeof(float));
    GPU_transpose(input,input_trans,batchSize,inChannels,numPoints);
    GPU_Bmm(input_trans,stn3d_out,bmm1_res,numPoints,inChannels,inChannels,encoderIC1,batchSize);
    GPU_transpose(bmm1_res,bmm1_res_trans,batchSize,numPoints,encoderIC1);
    GPU_CBR(batchSize,numPoints,encoderIC1,fstn_inChannel,dParams.featp.cb1,bmm1_res_trans,fstn_input);
    hipFree(input_trans);
    hipFree(bmm1_res);
    hipFree(bmm1_res_trans);

    std::cout << "PART3:STNkd"<< std::endl;
    int fstn_OC1 = 64;
    int fstn_OC2 = 128;
    int fstn_OC3 = 1024;
    int fstn_FC_OC1 = 512;
    int fstn_FC_OC2 = 256;
    int fstn_FC_OC3 = fstn_inChannel * fstn_inChannel ;
    float* fstn_CBR3_output;
    float* fstn_maxp_output;
    hipMalloc((void **)&fstn_CBR3_output, bn * fstn_OC3 * sizeof(float));
    hipMalloc((void **)&fstn_maxp_output, batchSize * fstn_OC3 * sizeof(float));
    GPU_CBR_3(fstn_OC1,fstn_OC2,fstn_OC3, batchSize, numPoints,fstn_inChannel,dParams.stnkdp.cb3, fstn_input, fstn_CBR3_output);   // conv-bn-relu * 3
    GPU_MaxPooling(fstn_OC3, batchSize, numPoints,fstn_CBR3_output, fstn_maxp_output); // Max pooling
    GPU_FBR_2_F(fstn_FC_OC1,fstn_FC_OC2,fstn_FC_OC3,batchSize,fstn_OC3,dParams.stnkdp.fb2f,fstn_maxp_output,stnkd_out);// fc-bn-relu * 2 + fc
    matrix_add_I(stnkd_out,64,batchSize);
    hipFree(fstn_CBR3_output);
    hipFree(fstn_maxp_output);

    std::cout << "PART4:TRANS->BMM->TRANS->CBR->CBM" << std::endl;
    int encoderOC2 = 128;
    float* fstn_input_trans;
    float* fstn_bmm1_res;
    float* fstn_bmm1_res_trans; // B C N
    float* cbr2_output;
    hipMalloc((void **)&fstn_input_trans, bn * fstn_inChannel * sizeof(float));
    hipMalloc((void **)&fstn_bmm1_res, batchSize*numPoints*fstn_inChannel * sizeof(float));
    hipMalloc((void **)&fstn_bmm1_res_trans, batchSize*fstn_inChannel*numPoints * sizeof(float));
    hipMalloc((void **)&cbr2_output, batchSize*encoderOC2*numPoints * sizeof(float));
    GPU_transpose(fstn_input,fstn_input_trans,batchSize,fstn_inChannel,numPoints);
    GPU_Bmm(fstn_input_trans,stnkd_out,fstn_bmm1_res,numPoints,fstn_inChannel,fstn_inChannel,fstn_inChannel,batchSize);
    GPU_transpose(fstn_bmm1_res,fstn_bmm1_res_trans,batchSize,numPoints,fstn_inChannel);
    GPU_CBR(batchSize,numPoints,fstn_inChannel,encoderOC2,dParams.featp.cb2,fstn_bmm1_res_trans,cbr2_output);
    //------CB MAX
    int encoderOC3 = 1024;
    int bnEOC3 = batchSize * numPoints * encoderOC3;
    //float* feat_conv3;
    float* feat_bn3;
    float* encoder_output;
    //hipMalloc((void **)&feat_conv3, bnEOC3 * sizeof(float));
    hipMalloc((void **)&feat_bn3, bnEOC3 * sizeof(float));
    hipMalloc((void **)&encoder_output, batchSize * encoderOC3 * sizeof(float));
    std::string convStr = "feat.conv3";
    std::string bnStr = "feat.bn3";
    CBWRAP_GPU(batchSize,numPoints,encoderOC2,encoderOC3,1,cbr2_output,
    dParams.featp.cb3.weight, dParams.featp.cb3.bias,
    dParams.featp.cb3.bn_weight, dParams.featp.cb3.bn_bias, 
    dParams.featp.cb3.bn_mean, dParams.featp.cb3.bn_var,feat_bn3);
    GPU_MaxPooling(encoderOC3, batchSize, numPoints,feat_bn3, encoder_output); // Max pooling
    
    hipFree(fstn_input_trans);
    hipFree(fstn_bmm1_res);
    hipFree(fstn_bmm1_res_trans); // B C N
    hipFree(cbr2_output);
    hipFree(feat_bn3);

    std::cout << "PART5:CLASSIFY" << std::endl;
    float* softmax_input;
    hipMalloc((void **)&softmax_input,sizeof(float)*batchSize*10);
    GPU_FBR_2_F(512,256,10,batchSize,encoderOC3,dParams.nonep,encoder_output,softmax_input,0);// fc-bn-relu * 2 + fc
    LogSoftMax_GPU(softmax_input, output, 10 , batchSize);
    std::cout << "----FINAL RESULT" << std::endl;
    std::vector<int> result(batchSize);

    std::vector<float> softmax_output_cpu(batchSize * 10);
    hipMemcpy(softmax_output_cpu.data(), output, batchSize * 10 * sizeof(float), hipMemcpyDeviceToHost);
    {
        for (int i = 0; i < batchSize; i++)
        {
            float max_value = softmax_output_cpu[i * 10];
            int max_index = 0;
            for (int j = 1; j < 10; j++)
            {
                if (softmax_output_cpu[i * 10 + j] > max_value)
                {
                    max_value = softmax_output_cpu[i * 10 + j];
                    max_index = j;
                }
            }
            result[i] = max_index;
        }
    }
    hipFree(softmax_input);
    return result;
}



std::vector<int> Inference_CPU (int inChannels,int batchSize,int numPoints,std::vector<float> input,std::vector<float> &output,std::vector<float> &stn3d_out,std::vector<float> &stnkd_out) {
    std::cout << "**********************START INFERENCE************************" << std::endl;
    std::cout << "PART1:STN3d" << std::endl;
    int bn = batchSize * numPoints;
    int OC1 = 64;
    int OC2 = 128;
    int OC3 = 1024;
    int FC_OC1 = 512;
    int FC_OC2 = 256;
    int FC_OC3 = 9;
    std::vector<float> CBR3_output(bn * OC3);
    std::vector<float> maxp_output(batchSize * OC3);
    //std::vector<float> FBR2F_output(batchSize * FC_OC3);
    CBR_3(OC1,OC2,OC3, batchSize, numPoints,inChannels,"feat.stn.", input, CBR3_output);   // conv-bn-relu * 3
    MaxPooling(OC3, batchSize, numPoints,CBR3_output, maxp_output); // Max pooling    
    FBR_2_F(FC_OC1,FC_OC2,FC_OC3,batchSize,OC3,"feat.stn.",maxp_output,stn3d_out);// fc-bn-relu * 2 + fc
    float I[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
    for (int i = 0; i < batchSize; ++i) {
        for (int j = 0; j < FC_OC3; ++j) {
            stn3d_out[i * FC_OC3 + j] += I[j];//batchSize * inic(3) * inic(3)
        }
    }

    std::cout << "PART2:TRANS->BMM->TRANS->CBR" << std::endl;
    int encoderIC1 = inChannels;
    int fstn_inChannel = 64;//encoderOC1
    std::vector<float> input_trans(bn * inChannels);
    std::vector<float> bmm1_res(batchSize*numPoints*encoderIC1);
    std::vector<float> bmm1_res_trans(batchSize*encoderIC1*numPoints);
    std::vector<float> fstn_input(batchSize*fstn_inChannel*numPoints);
    transpose(input,input_trans,batchSize,inChannels,numPoints);
    Bmm_cpu(input_trans,stn3d_out,bmm1_res,numPoints,inChannels,inChannels,encoderIC1,batchSize);
    transpose(bmm1_res,bmm1_res_trans,batchSize,numPoints,encoderIC1);
    CBR(1,batchSize,numPoints,encoderIC1,fstn_inChannel,"feat.",bmm1_res_trans,fstn_input);

    std::cout << "PART3:STNkd"<< std::endl;
    int fstn_OC1 = 64;
    int fstn_OC2 = 128;
    int fstn_OC3 = 1024;
    int fstn_FC_OC1 = 512;
    int fstn_FC_OC2 = 256;
    int fstn_FC_OC3 = fstn_inChannel * fstn_inChannel ;
    std::vector<float> fstn_CBR3_output(bn * fstn_OC3);
    std::vector<float> fstn_maxp_output(batchSize * fstn_OC3);
    //std::vector<float> fstn_FBR2F_output(batchSize * fstn_FC_OC3);
    CBR_3(fstn_OC1,fstn_OC2,fstn_OC3, batchSize, numPoints,fstn_inChannel,"feat.fstn.", fstn_input, fstn_CBR3_output);   // conv-bn-relu * 3
    MaxPooling(fstn_OC3, batchSize, numPoints,fstn_CBR3_output, fstn_maxp_output); // Max pooling
    FBR_2_F(fstn_FC_OC1,fstn_FC_OC2,fstn_FC_OC3,batchSize,fstn_OC3,"feat.fstn.",fstn_maxp_output,stnkd_out);// fc-bn-relu * 2 + fc
    for (int i = 0; i < batchSize; ++i) {
        for (int j = 0; j < fstn_FC_OC3; ++j) {
            stnkd_out[i * fstn_FC_OC3 + j] += (j % (fstn_inChannel + 1) == 0) ? 1.0f : 0.0f; //batchSize * 64 * 64
        }
    }

    std::cout << "PART4:TRANS->BMM->TRANS->CBR->CBM" << std::endl;
    int encoderOC2 = 128;
    std::vector<float> fstn_input_trans(bn * fstn_inChannel);
    std::vector<float> fstn_bmm1_res(batchSize * numPoints * fstn_inChannel);
    std::vector<float> fstn_bmm1_res_trans(batchSize * fstn_inChannel * numPoints); // B C N
    std::vector<float> cbr2_output(batchSize * encoderOC2 * numPoints);
    transpose(fstn_input,fstn_input_trans,batchSize,fstn_inChannel,numPoints);
    Bmm_cpu(fstn_input_trans,stnkd_out,fstn_bmm1_res,numPoints,fstn_inChannel,fstn_inChannel,fstn_inChannel,batchSize);
    transpose(fstn_bmm1_res,fstn_bmm1_res_trans,batchSize,numPoints,fstn_inChannel);
    CBR(2,batchSize,numPoints,fstn_inChannel,encoderOC2,"feat.",fstn_bmm1_res_trans,cbr2_output);
    //------CB MAX
    int encoderOC3 = 1024;
    int bnEOC3 = batchSize * numPoints * encoderOC3;
    std::vector<float> feat_conv3(bnEOC3);
    std::vector<float> feat_bn3(bnEOC3, 0);
    std::vector<float> encoder_output(batchSize * encoderOC3);
    std::string convStr = "feat.conv3";
    std::string bnStr = "feat.bn3";
    Conv1d_CPU(batchSize,numPoints,encoderOC2, encoderOC3, 1,cbr2_output, params[convStr + ".weight"], params[convStr + ".bias"], feat_conv3);
    BatchNorm1d_CPU(encoderOC3, batchSize, numPoints,params[bnStr + ".weight"], params[bnStr + ".bias"], params[RM(bnStr)], params[RV(bnStr)],feat_conv3,feat_bn3);
    MaxPooling(encoderOC3, batchSize, numPoints,feat_bn3, encoder_output); // Max pooling
    

    std::cout << "PART5:CLASSIFY" << std::endl;
    std::vector<float> softmax_input(batchSize*10);
    FBR_2_F(512,256,10,batchSize,encoderOC3,"",encoder_output,softmax_input,0);// fc-bn-relu * 2 + fc
    LogSoftMax_cpu(softmax_input, output, 10 , batchSize);
    std::cout << "----FINAL RESULT" << std::endl;
    std::vector<int> result(batchSize);
    {
        for (int i = 0; i < batchSize; i++)
        {
            float max_value = output[i * 10];
            int max_index = 0;
            for (int j = 1; j < 10; j++)
            {
                if (output[i * 10 + j] > max_value)
                {
                    max_value = output[i * 10 + j];
                    max_index = j;
                }
            }
            result[i] = max_index;
        }
    }
    return result;
}

// for TEST
void STN3d(float* x, int width, int batch_size, int ic, float* output,float* C1,float* C2,float* C3) { //x:batchsize*ic*N
    float epsilon = 1e-5;//默认的固定值

    // Define dimensions for each layer
    const int conv1_out_ics = 64;
    const int conv2_out_ics = 128;
    const int conv3_out_ics = 1024;
    const int fc1_out_features = 512;
    const int fc2_out_features = 256;

    // Temporary arrays for intermediate outputs
    std::vector<float> conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> conv2_out(batch_size * conv2_out_ics * width );
    std::vector<float> conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> bn_conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> bn_conv2_out(batch_size * conv2_out_ics * width );
    std::vector<float> bn_conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> relu_conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> relu_conv2_out(batch_size * conv2_out_ics * width );
    //std::vector<float> relu_conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> fc1_out(batch_size * fc1_out_features);
    std::vector<float> fc2_out(batch_size * fc2_out_features);
    std::vector<float> bn_fc1_out(batch_size * fc1_out_features);
    std::vector<float> bn_fc2_out(batch_size * fc2_out_features);
    std::vector<float> relu_fc1_out(batch_size * fc1_out_features);
    std::vector<float> relu_fc2_out(batch_size * fc2_out_features);
    //std::vector<float> max_pool_out(batch_size * conv3_out_ics);

    // Convolution layers
    Conv1d(batch_size,ic, conv1_out_ics, 1, width, x, params["feat.stn.conv1.weight"].data(), params["feat.stn.conv1.bias"].data(), conv1_out.data());
    // std::vector<float> x_vec(x, x + ic*batch_size*width);
    // std::vector<float> conv1_out_lt(batch_size * conv1_out_ics * width );
    // Conv1d_CPU(batch_size,width,ic,conv1_out_ics,1,x_vec,params["feat.stn.conv1.weight"],params["feat.stn.conv1.bias"],conv1_out_lt);
    // compareVectors(conv1_out,conv1_out_lt);

    // int iSize = ic*batch_size*width;
    // int oSize = batch_size * conv1_out_ics * width;
    // float* x_vec;
    // float* y_vec;
    // hipMalloc((void **)&x_vec,iSize*sizeof(float));
    // hipMalloc((void **)&y_vec,oSize*sizeof(float));
    // hipMemcpy(x_vec,x,iSize*sizeof(float),hipMemcpyHostToDevice);
    // Conv1d_GPU(batch_size,width,ic,conv1_out_ics,1,x_vec,params["feat.stn.conv1.weight"].data(),params["feat.stn.conv1.bias"].data(),y_vec);
    // compareVectors_GPU(conv1_out,y_vec,oSize);
    // printVector(conv1_out);
    // printVector_GPU(y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(y_vec);

    batchNorm1d(conv1_out.data(), params["feat.stn.bn1.weight"].data(), params["feat.stn.bn1.bias"].data(), bn_conv1_out.data(), params["feat.stn.bn1.running_mean"].data(),params["feat.stn.bn1.running_var"].data(), batch_size, width,conv1_out_ics, epsilon);
    
    // int iSize = ic*batch_size*width;
    // int oSize = batch_size * conv1_out_ics * width;
    // float* x_vec;
    // float* m_vec;
    // float* y_vec;
    // hipMalloc((void **)&x_vec,iSize*sizeof(float));
    // hipMalloc((void **)&m_vec,oSize*sizeof(float));
    // hipMalloc((void **)&y_vec,oSize*sizeof(float));
    // hipMemcpy(x_vec,x,iSize*sizeof(float),hipMemcpyHostToDevice);
    // Conv1d_GPU(batch_size,width,ic,conv1_out_ics,1,x_vec,params["feat.stn.conv1.weight"].data(),params["feat.stn.conv1.bias"].data(),m_vec);
    // BatchNorm1d_GPU(conv1_out_ics, batch_size, width,params["feat.stn.bn1.weight"].data(), params["feat.stn.bn1.bias"].data(), params["feat.stn.bn1.running_mean"].data(), params["feat.stn.bn1.running_var"].data(),m_vec,y_vec);
    // compareVectors_GPU(bn_conv1_out,y_vec,oSize);
    // printVector(conv1_out);
    // printVector_GPU(y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(y_vec);
    // hipFree(m_vec);
    // hipFree(m2_vec);
    
    relu(bn_conv1_out.data(), relu_conv1_out.data(), batch_size * conv1_out_ics * width );
    // std::vector<float> x_vec(x, x + ic*batch_size*width);
    // std::vector<float> relu_conv1_out_lt(batch_size * conv1_out_ics * width );
    // CBR(1,batch_size,width,ic,conv1_out_ics,"feat.stn.",x_vec,relu_conv1_out_lt);
    // compareVectors(relu_conv1_out_lt,relu_conv1_out);

    // int iSize = ic*batch_size*width;
    // int oSize = batch_size * conv1_out_ics * width;
    // float* x_vec;
    // float* m_vec;
    // float* m2_vec;
    // float* y_vec;
    // hipMalloc((void **)&x_vec,iSize*sizeof(float));
    // hipMalloc((void **)&m_vec,oSize*sizeof(float));
    // hipMalloc((void **)&m2_vec,oSize*sizeof(float));
    // hipMalloc((void **)&y_vec,oSize*sizeof(float));
    // hipMemcpy(x_vec,x,iSize*sizeof(float),hipMemcpyHostToDevice);
    // Conv1d_GPU(batch_size,width,ic,conv1_out_ics,1,x_vec,params["feat.stn.conv1.weight"].data(),params["feat.stn.conv1.bias"].data(),m_vec);
    // BatchNorm1d_GPU(conv1_out_ics, batch_size, width,params["feat.stn.bn1.weight"].data(), params["feat.stn.bn1.bias"].data(), params["feat.stn.bn1.running_mean"].data(), params["feat.stn.bn1.running_var"].data(),m_vec,m2_vec);
    // ReLU_GPU(batch_size,width,conv1_out_ics,m2_vec,y_vec);
    // compareVectors_GPU(relu_conv1_out,y_vec,oSize);
    // //printVector(conv1_out);
    // printVector_GPU(y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(y_vec);
    // hipFree(m_vec);
    // hipFree(m2_vec);

    // int iSize = ic * batch_size * width;
    // int oSize = batch_size * conv1_out_ics * width;
    // float *x_vec;
    // float *y_vec;
    // hipMalloc((void **)&x_vec, iSize * sizeof(float));
    // hipMalloc((void **)&y_vec, oSize * sizeof(float));
    // hipMemcpy(x_vec, x, iSize * sizeof(float), hipMemcpyHostToDevice);
    // GPU_CBR(1, batch_size, width, ic, conv1_out_ics, "feat.stn.", x_vec, y_vec,conv1_out,bn_conv1_out,relu_conv1_out,true);
    // compareVectors_GPU(relu_conv1_out, y_vec, oSize);
    // printVector_GPU(y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(y_vec);

    Conv1d(batch_size,conv1_out_ics, conv2_out_ics, 1, width , relu_conv1_out.data(), params["feat.stn.conv2.weight"].data(), params["feat.stn.conv2.bias"].data(), conv2_out.data());
    batchNorm1d(conv2_out.data(), params["feat.stn.bn2.weight"].data(), params["feat.stn.bn2.bias"].data(), bn_conv2_out.data(), params["feat.stn.bn2.running_mean"].data(),params["feat.stn.bn2.running_var"].data(), batch_size, width,conv2_out_ics, epsilon);
    relu(bn_conv2_out.data(), relu_conv2_out.data(), batch_size * conv2_out_ics * width );

    Conv1d(batch_size,conv2_out_ics, conv3_out_ics, 1, width , relu_conv2_out.data(), params["feat.stn.conv3.weight"].data(), params["feat.stn.conv3.bias"].data(),conv3_out.data());
    batchNorm1d(conv3_out.data(), params["feat.stn.bn3.weight"].data(), params["feat.stn.bn3.bias"].data(), bn_conv3_out.data(),params["feat.stn.bn3.running_mean"].data(),params["feat.stn.bn3.running_var"].data(), batch_size, width,conv3_out_ics, epsilon);
    relu(bn_conv3_out.data(), C1, batch_size * conv3_out_ics * width );
    // std::vector<float> x_vec(x, x + ic*batch_size*width);
    // std::vector<float> relu_conv3_out_lt(batch_size * conv3_out_ics * width );
    // CBR_3(64,128,1024,batch_size,width,ic,"feat.stn.",x_vec,relu_conv3_out_lt);
    //compareVectors(relu_conv3_out_lt,relu_conv3_out);
    // GPU-CHECK
    // int iSize = ic * batch_size * width;
    // int oSize = batch_size * conv3_out_ics * width;
    // float *x_vec;
    // float *y_vec;
    // hipMalloc((void **)&x_vec, iSize * sizeof(float));
    // hipMalloc((void **)&y_vec, oSize * sizeof(float));
    // hipMemcpy(x_vec, x, iSize * sizeof(float), hipMemcpyHostToDevice);
    // GPU_CBR_3(64,128,1024,batch_size,width,ic,"feat.stn.",x_vec,y_vec);
    // compareVectors_GPU(relu_conv3_out,y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(y_vec);

    // Max pooling
    max_along_dim(C1, C2, batch_size, conv3_out_ics, width);
    // std::vector<float> max_pool_out_LT(batch_size * conv3_out_ics);
    // MaxPooling(conv3_out_ics,batch_size,width,relu_conv3_out,max_pool_out_LT);
    // compareVectors(max_pool_out,max_pool_out_LT);

    // Fully connected layers
    FullConnect(batch_size, conv3_out_ics, fc1_out_features, C2, params["feat.stn.fc1.weight"].data(), fc1_out.data(), params["feat.stn.fc1.bias"].data());
    batchNorm1d(fc1_out.data(), params["feat.stn.bn4.weight"].data(), params["feat.stn.bn4.bias"].data(), bn_fc1_out.data(), params["feat.stn.bn4.running_mean"].data(),params["feat.stn.bn4.running_var"].data(), batch_size, 1,fc1_out_features, epsilon);
    relu(bn_fc1_out.data(), relu_fc1_out.data(), batch_size * fc1_out_features);

    FullConnect(batch_size, fc1_out_features, fc2_out_features, relu_fc1_out.data(), params["feat.stn.fc2.weight"].data(), fc2_out.data(), params["feat.stn.fc2.bias"].data());
    batchNorm1d(fc2_out.data(), params["feat.stn.bn5.weight"].data(), params["feat.stn.bn5.bias"].data(), bn_fc2_out.data(), params["feat.stn.bn5.running_mean"].data(),params["feat.stn.bn5.running_var"].data(), batch_size, 1,fc2_out_features, epsilon);
    relu(bn_fc2_out.data(), relu_fc2_out.data(), batch_size * fc2_out_features);

    FullConnect(batch_size, fc2_out_features, ic*ic, relu_fc2_out.data(), params["feat.stn.fc3.weight"].data(), C3, params["feat.stn.fc3.bias"].data());
    // std::vector<float> output_lt(batch_size * ic*ic);
    // std::vector<float> outvec(output, output+batch_size*ic*ic);
    // FBR_2_F(fc1_out_features,fc2_out_features,ic*ic,batch_size,conv3_out_ics,"feat.stn.",max_pool_out,output_lt);
    // compareVectors(outvec,output_lt);
    // GPU-CHECK:MAXP -> FBR2-F
    // int iSize = batch_size * conv3_out_ics * width;
    // int mSize = batch_size * conv3_out_ics;
    // int oSize = batch_size * ic*ic;
    // float *x_vec;
    // float *m_vec;
    // float *y_vec;
    // hipMalloc((void **)&x_vec, iSize * sizeof(float));
    // hipMalloc((void **)&m_vec, mSize * sizeof(float));
    // hipMalloc((void **)&y_vec, oSize * sizeof(float));
    // hipMemcpy(x_vec, relu_conv3_out.data(), iSize * sizeof(float), hipMemcpyHostToDevice);
    // GPU_MaxPooling(conv3_out_ics,batch_size,width,x_vec,m_vec);
    // compareVectors_GPU(max_pool_out,m_vec,mSize);
    // GPU_FBR_2_F(fc1_out_features,fc2_out_features,ic*ic,batch_size,conv3_out_ics,"feat.stn.",m_vec,y_vec,
    // 3,relu_fc1_out,relu_fc2_out,false);
    // compareVectors_GPU_float(output,y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(m_vec);
    // hipFree(y_vec);


    // Add identity matrix
    float identity[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
    for (int i = 0; i < batch_size; ++i) {
        for (int j = 0; j < 9; ++j) {
            output[i * 9 + j] = C3[i*9+j]+identity[j];
        }
    }
}
void STNkd(float* x, int width, int batch_size, int ic, float* output) {
    float epsilon = 1e-5;//默认的固定值

    // Define dimensions for each layer
    const int conv1_out_ics = 64;
    const int conv2_out_ics = 128;
    const int conv3_out_ics = 1024;
    const int fc1_out_features = 512;
    const int fc2_out_features = 256;

    // Temporary arrays for intermediate outputs
    std::vector<float> conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> conv2_out(batch_size * conv2_out_ics * width );
    std::vector<float> conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> bn_conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> bn_conv2_out(batch_size * conv2_out_ics * width );
    std::vector<float> bn_conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> relu_conv1_out(batch_size * conv1_out_ics * width);
    std::vector<float> relu_conv2_out(batch_size * conv2_out_ics * width );
    std::vector<float> relu_conv3_out(batch_size * conv3_out_ics * width );
    std::vector<float> fc1_out(batch_size * fc1_out_features);
    std::vector<float> fc2_out(batch_size * fc2_out_features);
    std::vector<float> bn_fc1_out(batch_size * fc1_out_features);
    std::vector<float> bn_fc2_out(batch_size * fc2_out_features);
    std::vector<float> relu_fc1_out(batch_size * fc1_out_features);
    std::vector<float> relu_fc2_out(batch_size * fc2_out_features);
    std::vector<float> max_pool_out(batch_size * conv3_out_ics);
    
    // Convolution layers
    Conv1d(batch_size,ic, conv1_out_ics, 1, width, x, params["feat.fstn.conv1.weight"].data(),  params["feat.fstn.conv1.bias"].data(),conv1_out.data());
    batchNorm1d(conv1_out.data(), params["feat.fstn.bn1.weight"].data(), params["feat.fstn.bn1.bias"].data(), bn_conv1_out.data(), params["feat.fstn.bn1.running_mean"].data(),params["feat.fstn.bn1.running_var"].data(), batch_size, width, conv1_out_ics, epsilon);
    relu(bn_conv1_out.data(), relu_conv1_out.data(), batch_size * conv1_out_ics * width );

    Conv1d(batch_size,conv1_out_ics, conv2_out_ics, 1, width , relu_conv1_out.data(), params["feat.fstn.conv2.weight"].data(), params["feat.fstn.conv2.bias"].data(),conv2_out.data());
    batchNorm1d(conv2_out.data(), params["feat.fstn.bn2.weight"].data(), params["feat.fstn.bn2.bias"].data(), bn_conv2_out.data(), params["feat.fstn.bn2.running_mean"].data(),params["feat.fstn.bn2.running_var"].data(), batch_size, width, conv2_out_ics, epsilon);
    relu(bn_conv2_out.data(), relu_conv2_out.data(), batch_size * conv2_out_ics * width );

    Conv1d(batch_size,conv2_out_ics, conv3_out_ics, 1, width , relu_conv2_out.data(), params["feat.fstn.conv3.weight"].data(), params["feat.fstn.conv3.bias"].data(),conv3_out.data());
    batchNorm1d(conv3_out.data(), params["feat.fstn.bn3.weight"].data(), params["feat.fstn.bn3.bias"].data(), bn_conv3_out.data(),params["feat.fstn.bn3.running_mean"].data(),params["feat.fstn.bn3.running_var"].data(), batch_size, width, conv3_out_ics, epsilon);
    relu(bn_conv3_out.data(), relu_conv3_out.data(), batch_size * conv3_out_ics * width );

    // Max pooling
    max_along_dim(relu_conv3_out.data(), max_pool_out.data(), batch_size, conv3_out_ics,width);

    // Fully connected layers
    FullConnect(batch_size, conv3_out_ics, fc1_out_features, max_pool_out.data(), params["feat.fstn.fc1.weight"].data(), fc1_out.data(), params["feat.fstn.fc1.bias"].data());
    batchNorm1d(fc1_out.data(), params["feat.fstn.bn4.weight"].data(), params["feat.fstn.bn4.bias"].data(), bn_fc1_out.data(), params["feat.fstn.bn4.running_mean"].data(),params["feat.fstn.bn4.running_var"].data(), batch_size, 1,fc1_out_features, epsilon);
    relu(bn_fc1_out.data(), relu_fc1_out.data(), batch_size * fc1_out_features);

    FullConnect(batch_size, fc1_out_features, fc2_out_features, relu_fc1_out.data(), params["feat.fstn.fc2.weight"].data(), fc2_out.data(), params["feat.fstn.fc2.bias"].data());
    batchNorm1d(fc2_out.data(), params["feat.fstn.bn5.weight"].data(), params["feat.fstn.bn5.bias"].data(), bn_fc2_out.data(), params["feat.fstn.bn5.running_mean"].data(),params["feat.fstn.bn5.running_var"].data(), batch_size, 1,fc2_out_features, epsilon);
    relu(bn_fc2_out.data(), relu_fc2_out.data(), batch_size * fc2_out_features);

    FullConnect(batch_size, fc2_out_features, ic*ic, relu_fc2_out.data(), params["feat.fstn.fc3.weight"].data(), output, params["feat.fstn.fc3.bias"].data());

    for (int i = 0; i < batch_size; ++i) {
        for (int j = 0; j < ic * ic; ++j) {
            output[i * (ic * ic) + j] += (j % (ic + 1) == 0) ? 1.0f : 0.0f; // 适应 ic 数量
        }
    }
}
void PointNetEncoder(float* x, int batch_size, int ic, int N, float* trans, float* trans_feat, float* final_x,float* C1,float* C2,float* C3) {
    float epsilon = 1e-5;//默认的固定值

    const int conv1_out_ics = 64;
    const int conv2_out_ics = 128;
    const int conv3_out_ics = 1024;
    std::vector<float> conv1_out(batch_size * conv1_out_ics * N);
    std::vector<float> conv2_out(batch_size * conv2_out_ics * N);
    std::vector<float> relu_conv1_out(batch_size * conv1_out_ics * N);
    std::vector<float> relu_conv2_out(batch_size * conv2_out_ics * N);
    std::vector<float> conv3_out(batch_size * conv3_out_ics * N);
    std::vector<float> bn_conv1_out(batch_size * conv1_out_ics * N);
    std::vector<float> bn_conv2_out(batch_size * conv2_out_ics * N);
    std::vector<float> bn_conv3_out(batch_size * conv3_out_ics * N);
    //std::vector<float> max_pool_out(batch_size * conv3_out_ics);
    std::vector<float> x_trans(batch_size * ic * N);
    std::vector<float> x_trans_trans(batch_size * ic * N);
    std::vector<float> x_trans_trans_trans(batch_size * conv1_out_ics * N);
    std::vector<float> x_trans_trans_trans_mul_trans_feat(batch_size * conv1_out_ics * N);
    // Apply STN3d
    STN3d(x, N, batch_size, ic, trans,C1,C2,C3); //trans:batchsize*ic*ic
    
    // Transpose input data: [B, C, N] -> [B, N, C]
    transpose_xtf(x, x_trans.data(), batch_size, ic, N); //x_trans:batchsize*N*ic
    std::vector<float> x_trans_mul_trans(batch_size * ic * N );
    bmm(x_trans.data(), trans, x_trans_mul_trans.data(), batch_size, N, ic, ic); //x_trans_mul_trans:batchsize*N*ic

    // int iSize = ic*batch_size*N;
    // int i1Size = ic*ic*batch_size;
    // int oSize = ic*batch_size*N;
    // float* x_vec;
    // float* x1_vec;
    // float* y_vec;
    // hipMalloc((void **)&x_vec,iSize*sizeof(float));
    // hipMalloc((void **)&x1_vec,i1Size*sizeof(float));
    // hipMalloc((void **)&y_vec,oSize*sizeof(float));
    // hipMemcpy(x_vec,x_trans.data(),iSize*sizeof(float),hipMemcpyHostToDevice);
    // hipMemcpy(x1_vec,trans,i1Size*sizeof(float),hipMemcpyHostToDevice);
    // GPU_Bmm(x_vec,x1_vec,y_vec,N,ic,ic,ic,batch_size);
    // compareVectors_GPU(x_trans_mul_trans,y_vec,oSize);
    // hipFree(x_vec);
    // hipFree(x1_vec);
    // hipFree(y_vec);

    transpose_xtf(x_trans_mul_trans.data(), x_trans_trans.data(), batch_size, N, ic);//x_trans_trans:batchsize*ic*N

    Conv1d(batch_size,ic, conv1_out_ics, 1, N, x_trans_trans.data(), params["feat.conv1.weight"].data(),params["feat.conv1.bias"].data(), conv1_out.data());
    batchNorm1d(conv1_out.data(), params["feat.bn1.weight"].data(), params["feat.bn1.bias"].data(), bn_conv1_out.data(), params["feat.bn1.running_mean"].data(),params["feat.bn1.running_var"].data(), batch_size, N, conv1_out_ics, epsilon);
    relu(bn_conv1_out.data(), relu_conv1_out.data(), batch_size * conv1_out_ics * N );//conv1_out:batch_size * conv1_out_ics * N

    STNkd(relu_conv1_out.data(), N, batch_size, conv1_out_ics, trans_feat);//trans_feat:batchsize*conv1_out_ics*conv1_out_ics
    transpose_xtf(relu_conv1_out.data(), x_trans_trans_trans.data(), batch_size, conv1_out_ics, N);// x_trans_trans_trans:batchsize*N*conv1_out_ics
    bmm(x_trans_trans_trans.data(), trans_feat, x_trans_trans_trans_mul_trans_feat.data(), batch_size, N, conv1_out_ics, conv1_out_ics); //
    transpose_xtf(x_trans_trans_trans_mul_trans_feat.data(), x_trans_trans_trans.data(), batch_size, N, conv1_out_ics);

    Conv1d(batch_size,conv1_out_ics, conv2_out_ics, 1, N, x_trans_trans_trans.data(), params["feat.conv2.weight"].data(), params["feat.conv2.bias"].data(),conv2_out.data());
    batchNorm1d(conv2_out.data(), params["feat.bn2.weight"].data(), params["feat.bn2.bias"].data(), bn_conv2_out.data(), params["feat.bn2.running_mean"].data(),params["feat.bn2.running_var"].data(), batch_size, N, conv2_out_ics, epsilon);
    relu(bn_conv2_out.data(), relu_conv2_out.data(), batch_size * conv2_out_ics * N );//conv2_out.data():batch_size * conv2_out_ics * N

    Conv1d(batch_size,conv2_out_ics, conv3_out_ics, 1, N, relu_conv2_out.data(), params["feat.conv3.weight"].data(),params["feat.conv3.bias"].data(), conv3_out.data());
    batchNorm1d(conv3_out.data(), params["feat.bn3.weight"].data(), params["feat.bn3.bias"].data(), bn_conv3_out.data(), params["feat.bn3.running_mean"].data(),params["feat.bn3.running_var"].data(), batch_size, N,  conv3_out_ics, epsilon);
   
    max_along_dim(bn_conv3_out.data(), final_x, batch_size, conv3_out_ics,N);//final_x:batchsize*1024(conv3_out_ics)
}
std::vector<int> get_model(float* x, int batch_size, int ic, int N, float* trans, float* trans_feat, float* final_x,float* C1,float* C2,float* C3){
    float epsilon = 1e-5;//默认的固定值
    const int fc1_in_features = 1024;
    const int fc1_out_features = 512;
    const int fc2_out_features = 256;
    const int fc3_out_features = 10;
    std::vector<float> fc1_out(batch_size * fc1_out_features);
    std::vector<float> fc2_out(batch_size * fc2_out_features);
    std::vector<float> bn_fc1_out(batch_size * fc1_out_features);
    std::vector<float> bn_fc2_out(batch_size * fc2_out_features);
    std::vector<float> fc3_out(batch_size * fc3_out_features);
    std::vector<float> fc1_in(batch_size * fc1_in_features);
    std::vector<float> relu_fc1_out(batch_size * fc1_out_features);
    std::vector<float> relu_fc2_out(batch_size * fc2_out_features);
    
    PointNetEncoder(x,batch_size,ic,N, trans, trans_feat, fc1_in.data(),C1,C2,C3);
    
    FullConnect(batch_size, fc1_in_features, fc1_out_features, fc1_in.data(), params["fc1.weight"].data(), fc1_out.data(), params["fc1.bias"].data());
    batchNorm1d(fc1_out.data(), params["bn1.weight"].data(), params["bn1.bias"].data(), bn_fc1_out.data(), params["bn1.running_mean"].data(),params["bn1.running_var"].data(), batch_size, 1, fc1_out_features, epsilon);
    relu(bn_fc1_out.data(), relu_fc1_out.data(), batch_size * fc1_out_features);

    FullConnect(batch_size, fc1_out_features, fc2_out_features, relu_fc1_out.data(), params["fc2.weight"].data(), fc2_out.data(), params["fc2.bias"].data());
    batchNorm1d(fc2_out.data(), params["bn2.weight"].data(), params["bn2.bias"].data(), bn_fc2_out.data(), params["bn2.running_mean"].data(),params["bn2.running_var"].data(), batch_size, 1, fc2_out_features, epsilon);
    relu(bn_fc2_out.data(), relu_fc2_out.data(), batch_size * fc2_out_features);

    FullConnect(batch_size, fc2_out_features, fc3_out_features, relu_fc2_out.data(), params["fc3.weight"].data(), fc3_out.data(), params["fc3.bias"].data());//batchsize*fc3_out_features

    log_softmax(fc3_out.data(), final_x, batch_size, fc3_out_features);

    std::vector<int> result(batch_size);
        for(int b = 0; b <  batch_size; ++b)
        {
            int max_index=0;
            float max=-FLT_MAX;
            for(int index=0;index<10;index++)
            {
                if(final_x[b*10+index]>max)
                {
                    max_index = index;
                    max = final_x[b*10+index];
                }
            }
            result[b]=max_index;
        }
        return result;
}

int main(int argc, char *argv[]) {
    
    // 读取模型参数
    std::string dir = argv[1]; 
    read_params(dir);

    // 读取训练集数据
    std::string file_path = "./data/test_point_clouds.h5";
    std::vector<std::vector<float>> list_of_points;
    std::vector<int> list_of_labels;
    read_h5_file(file_path, list_of_points, list_of_labels);

    // 开始计时，使用chrono计时，不支持其它计时方式
    auto start = std::chrono::high_resolution_clock::now();
    int batchSize = 4;
    int ic = 3;
    int correct_num =0;

    //迁移参数
    read_stndP("feat.stn.", dParams.stn3dp);
    read_stndP("feat.fstn.", dParams.stnkdp);
    read_CB3P("feat.", dParams.featp);
    read_FB2FP("", dParams.nonep, 0);

    // 开始计时，使用chrono计时，不支持其它计时方式
    std::cout << "total :" << list_of_labels.size() << std::endl;
    for (size_t i = 0; i < list_of_points.size(); i+=batchSize) {

        std::cout << "ITERATION: " << i << ": ";

        //当前循环BATCHSIZE
        size_t curB = (batchSize < list_of_points.size() - i) ? batchSize : list_of_points.size() - i;
        
        //当前循环中NUMPOINTS最少的点
        int np = list_of_points[i].size() / ic;
        for (int j = 0; j < curB; j++) 
        {
            np = (list_of_points[i + j].size() / ic < np) ? list_of_points[i + j].size() / ic : np;
        }

        std::cout << "CUTOFF INPUT: " << i << "np is : " << np <<std::endl;;
        std::vector<float> input(curB * np * ic);
        std::vector<float> trans(curB * ic * ic);
        std::vector<float> trans_feat(curB * 64 * 64);
        std::vector<float> final_x(curB * 10);
        for (int b = 0; b < curB; ++b)
        {
            for (int w = 0; w < np; ++w)
            {
                for (int c = 0; c < ic; ++c)
                {
                    input[b * np * ic + w * ic + c] = list_of_points[i + b][w * ic + c];
                }
            }
        }

        //输入输出
        std::vector<float> input_trans(curB * np * ic);
        transpose(input, input_trans, curB, np, ic);
        std::vector<int> result(curB,0);

        //推理与结果
        int transSize = curB * ic * ic;
        int transFeatSize = curB * 64 * 64;
        float *device_input_trans;
        hipMalloc((void **)&device_input_trans, curB * np * ic * sizeof(float));
        hipMemcpy(device_input_trans, input_trans.data(), curB * np * ic * sizeof(float), hipMemcpyHostToDevice);
        float *trans_lt_gpu;
        float *trans_feat_lt_gpu;
        float *netOut_gpu;
        hipMalloc((void **)&trans_lt_gpu, transSize * sizeof(float));
        hipMalloc((void **)&trans_feat_lt_gpu, transFeatSize * sizeof(float));
        hipMalloc((void **)&netOut_gpu, curB * 10 * sizeof(float));
        result = Inference_GPU(ic,curB,np,device_input_trans,netOut_gpu,trans_lt_gpu,trans_feat_lt_gpu);
        //result=Inference_CPU(ic,curB,np, input_trans,final_x,trans,trans_feat);
        for (int b = 0; b < curB; ++b)
        {
            correct_num += (result[b] == list_of_labels[i + b]);
            if (result[b] == list_of_labels[i + b])
                std::cout << (i + b) << std::endl;
        }
        std::cout << "END INFERENCE:: iter :" << i << " correct_num :" << correct_num << " iter_batchsize :" << curB << std::endl;
        std::cout << "total :" << list_of_labels.size() << std::endl;
        printVector<int> (result);
    }
    std::cout << "total :" << list_of_labels.size() << std::endl;
    std::cout << "correct_num :" << correct_num << std::endl;
	float correct_rate = (float)correct_num/(float)list_of_labels.size();
    freeDP(dParams);
	// 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
	hipDeviceSynchronize();

    // 结束计时
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    std::cout << std::fixed << std::setprecision(4) << diff.count() << correct_rate;

    return 0;
}

// int main(int argc, char *argv[]) {
    
//     // 读取模型参数
//     std::string dir = "./params/150epoch";  
//     read_params(dir);

//     // 读取训练集数据：此处无用
//     std::string file_path = "./data/test_point_clouds.h5";
//     std::vector<std::vector<float>> list_of_points;
//     std::vector<int> list_of_labels;
//     read_h5_file(file_path, list_of_points, list_of_labels);

//     //设定参数：ic*b*np
//     int ic = 3;
//     int b = 32;
//     int np = 13;
//     int transSize = b * ic *ic;
//     int transFeatSize = b * 64 * 64; 
//     std::vector<float> input(ic * b * np);
//     //基准
//     std::vector<float> trans(b * ic * ic);
//     std::vector<float> trans_feat(b * 64 * 64);
//     std::vector<float> final_y(b * 10);
//     std::vector<int> result;
//     std::vector<int> result_lt;
//     //待验证:REVISION
//     // std::vector<float> trans_lt(b * ic * ic);
//     // std::vector<float> trans_feat_lt(b * 64 * 64);
//     // std::vector<float> netOut(b * 10);
//     float *trans_lt_gpu;
//     float *trans_feat_lt_gpu;
//     float *netOut_gpu;
//     hipMalloc((void **)&trans_lt_gpu, transSize * sizeof(float));
//     hipMalloc((void **)&trans_feat_lt_gpu, transFeatSize * sizeof(float));
//     hipMalloc((void **)&netOut_gpu, b * 10 * sizeof(float));

//     // 生成输入
//     std::random_device rd;  
//     std::mt19937 eng(rd()); 
//     std::uniform_real_distribution<float> distr(0.0f, 1.0f); 
//     for (auto& value : input) {
//         value = distr(eng); 
//     }
//     std::vector<float> input_trans(b * np  * ic);
//     transpose(input,input_trans,b,np,ic );

//     //推理:REVISION
//     std::vector<float> C1(b * np * 1024);
//     std::vector<float> C2(b * 1024);
//     std::vector<float> C3(b * 9);
//     result=get_model(input_trans.data(), b, ic, np, trans.data(), trans_feat.data(), final_y.data(),C1.data(),C2.data(),C3.data());

//     float* device_input_trans;
//     hipMalloc((void **)&device_input_trans, b * np  * ic * sizeof(float));
//     hipMemcpy(device_input_trans, input_trans.data(), b * np  * ic * sizeof(float), hipMemcpyHostToDevice);
//     result_lt=Inference_GPU(ic, b, np, device_input_trans, netOut_gpu, trans_lt_gpu, trans_feat_lt_gpu,C1,C2,C3,trans,false);
//     //result_lt=Inference_CPU(ic, b, np, input_trans, netOut_gpu, trans_lt, trans_feat_lt);
    

//     //对比结果
//     compareVectors_GPU(trans,trans_lt_gpu,transSize);
//     compareVectors_GPU(trans_feat,trans_feat_lt_gpu,transFeatSize);
//     hipFree(trans_lt_gpu);
//     hipFree(trans_feat_lt_gpu);
//     hipFree(netOut_gpu);
//     //对比结果
//     compareVectors<int>(result_lt,result);
//     for (const auto& value : result) {
//         std::cout << value << " ";
//     }
//     std::cout << std::endl;
//     for (const auto& value : result_lt) {
//         std::cout << value << " ";
//     }
//     std::cout << std::endl;
//     return 0;
// }

